#include <stdio.h>
#include <hipsparse.h>
#include <hipsolver.h>

#include "herramientas.h"
#include "sistema_de_prueba.h"

#define VALUE 256

const int ROWS = VALUE;
const int COLUMNS = VALUE;
const int VECTOR_SIZE = VALUE;

int main(int argc, char const *argv[])
{
    double *A = NULL;
    double *b = NULL;
    double *csrValA = NULL;
    int *csrRowPtrA = NULL;
    int *csrColIndA = NULL;

    int N = ROWS;
    int NNZA;
    double TOL = 1e-6;
    int REORDER = 0;
    double *x = NULL;
    int singularity;

    A = allocate_memory_matrix(ROWS, COLUMNS);
    b = allocate_memory_vector(VECTOR_SIZE);
    x = allocate_memory_vector(VECTOR_SIZE);
    crear_matriz_de_prueba(A, ROWS, COLUMNS);
    crear_vector_de_prueba(b, VECTOR_SIZE);
    NNZA = obtener_nnzMatriz(A, ROWS, COLUMNS);

    csrValA = allocate_memory_vector(NNZA);
    csrColIndA = allocate_memory_vector_integer(NNZA);
    csrRowPtrA = allocate_memory_vector_integer(ROWS + 1);
    obtener_formato_csr(A, ROWS, COLUMNS, csrValA, csrRowPtrA, csrColIndA);

    // Variables CUDA
    hipsolverSpHandle_t handle = NULL;
    hipsparseMatDescr_t descrA = NULL;
    double *d_b = NULL;
    double *d_csrValA = NULL;
    int *d_csrRowPtrA = NULL;
    int *d_csrColIndA = NULL;
    double *d_x = NULL;

    d_b = allocateMemoryVectorDevice(VECTOR_SIZE);
    d_x = allocateMemoryVectorDevice(VECTOR_SIZE);
    d_csrValA = allocateMemoryVectorDevice(NNZA);
    d_csrColIndA = allocateMemoryVectorDeviceInteger(NNZA);
    d_csrRowPtrA = allocateMemoryVectorDeviceInteger(ROWS + 1);

    hipsolverSpCreate(&handle);
    hipsparseCreateMatDescr(&descrA);
    hipsparseSetMatType(descrA, HIPSPARSE_MATRIX_TYPE_GENERAL); hipsparseSetMatIndexBase(descrA, HIPSPARSE_INDEX_BASE_ZERO);

    transfer_vector_host_to_device(b, d_b, VECTOR_SIZE);
    transfer_vector_host_to_device(x, d_x, VECTOR_SIZE);
    transfer_vector_host_to_device(csrValA, d_csrValA, NNZA);
    transfer_vector_host_to_device_integer(csrColIndA, d_csrColIndA, NNZA);
    transfer_vector_host_to_device_integer(csrRowPtrA, d_csrRowPtrA, ROWS + 1);

    // cusolverSpDcsrlsvluHost(handle, N, NNZA, descrA, csrValA, csrRowPtrA, csrColIndA, b, TOL, REORDER, x, &singularity);
    hipsolverSpDcsrlsvqr(handle, N, NNZA, descrA, d_csrValA, d_csrRowPtrA, d_csrColIndA, d_b, TOL, REORDER, d_x, &singularity);
    // hipsolverSpDcsrlsvchol(handle,N, NNZA, descrA, d_csrValA, d_csrRowPtrA, d_csrColIndA, d_b, TOL, REORDER, d_x, &singularity);

    transfer_vector_device_to_host(d_x, x, VECTOR_SIZE);

    hipDeviceSynchronize();

    printf("n = %d\n", singularity);
    print_vector(x, VECTOR_SIZE);

    hipFree(d_csrRowPtrA);
    hipFree(d_csrColIndA);
    hipFree(d_csrValA);
    hipFree(d_x);
    hipFree(d_b);

    free(x);
    free(csrColIndA);
    free(csrRowPtrA);
    free(csrValA);
    free(b);
    free(A);

    return 0;
}
